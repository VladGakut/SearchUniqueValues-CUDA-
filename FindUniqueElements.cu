
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <random>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <thrust/unique.h>

#define MILLION             1'000'000
#define NUMBER_OF_ELEMENTS  10 * MILLION
#define NUMBER_OF_UNIQUE    1'000


int main(void)
{
    thrust::host_vector<int> h_vec(NUMBER_OF_ELEMENTS);

    std::random_device rd;
    thrust::default_random_engine rng(rd());
    thrust::uniform_int_distribution<int> dist(0, NUMBER_OF_UNIQUE);

    thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

    thrust::device_vector<int> d_vec = h_vec;

    thrust::sort(d_vec.begin(), d_vec.end());
    auto last = thrust::unique(d_vec.begin(), d_vec.end());

    std::cout << "Unique values: ";
    for (auto it = d_vec.begin(); it != last; ++it)
    {
        std::cout << *it << " ";
    }

    return 0;
}
